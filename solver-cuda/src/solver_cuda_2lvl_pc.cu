#include "hip/hip_runtime.h"
/*
 * mbsolve: Framework for solving the Maxwell-Bloch/-Lioville equations
 *
 * Copyright (c) 2016, Computational Photonics Group, Technical University of
 * Munich.
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301  USA
 */

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <common_cuda.hpp>
#include <solver_cuda_2lvl_pc.hpp>

namespace mbsolve {

static solver_factory<solver_cuda_2lvl_pc> factory("cuda-2lvl-pc");

/* initialize memory kernel */
__global__ void init_memory(real *d, real *e, real *h, unsigned int *indices)
{
    unsigned int gsize = blockDim.x * gridDim.x;
    unsigned int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int mat_idx = indices[gidx];

    /* TODO: alternative initializations */
    hiprandState_t rand_state;

    /* initialize random number generator */
    hiprand_init(clock64(), gidx, 0, &rand_state);

    if (gidx == blockDim.x * gridDim.x - 1) {
	h[gidx + 1] = 0.0;
    }
    h[gidx] = 0.0;
    e[gidx] = 0.0;
    //   e[gidx] = hiprand_uniform(&rand_state) * 1e-15;

    d[gsize * 0 + gidx] = l_sim_consts[mat_idx].inversion_init;
    d[gsize * 1 + gidx] = 0.0;
    d[gsize * 2 + gidx] = 0.0;
}

__global__ void makestep_h(const real *ge, real *gh, unsigned int *indices)
{
    int idx = threadIdx.x;
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int mat_idx = indices[gidx];

    extern __shared__ real e[];

    if ((idx == 0) && (gidx != 0)) {
	e[idx] = ge[gidx - 1];
    }
    e[idx + 1] = ge[gidx];

    __syncthreads();

    /* TODO: alternative boundary conditions? */
    /* TODO: different kernel or templated version?? */
    /* open circuit boundary conditions already set */
    /* gh_ghz[0] = 0; */
    /* gh_ghz[N_x] = 0; */

    if (gidx != 0) {
	gh[gidx] += l_sim_consts[mat_idx].M_CH * (e[idx + 1] - e[idx]);
    }
}


__global__ void makestep_e_dm(real *d, const real *gh, real *ge,
			      unsigned int *indices, real *scratch, real *sd,
                              unsigned int source_ct, unsigned int copy_ct,
                              unsigned int n)
{
    int gsize = blockDim.x * gridDim.x;
    int size = blockDim.x;
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = threadIdx.x;
    int mat_idx = indices[gidx];

    extern __shared__ real h[];
    real *inv = &h[size + 1];
    real *dm12i = &h[2 * size + 1];
    real *dm12r = &h[3 * size + 1];
    real *e = &h[5 * size + 1];

    h[idx] = gh[gidx];
    if (idx == blockDim.x - 1) {
	h[idx + 1] = gh[gidx + 1];
    }
    inv[idx] = d[gsize * 0 + gidx];
    dm12i[idx] = d[gsize * 1 + gidx];
    dm12r[idx] = d[gsize * 2 + gidx];
    e[idx] = ge[gidx];

    __syncthreads();

    real inv_e = inv[idx];
    real dm12i_e = dm12i[idx];
    real dm12r_e = dm12r[idx];
    real e_e = e[idx];

    /* execute prediction - correction steps */
    for (int pc_step = 0; pc_step < 4; pc_step++) {

	real inversion = 0.5 * (inv[idx] + inv_e);
	real rho12i = 0.5 * (dm12i[idx] + dm12i_e);
	real rho12r = 0.5 * (dm12r[idx] + dm12r_e);
        real field = 0.5 * (e[idx] + e_e);
	real OmRabi = l_sim_consts[mat_idx].d12 * field;

	/* dm11 */
	inv_e = inv[idx] + l_sim_consts[mat_idx].d_t *
	    (- 4.0 * OmRabi * rho12i
             - l_sim_consts[mat_idx].tau1 *
             (inversion - l_sim_consts[mat_idx].equi_inv));

	/* imag dm12 */
	dm12i_e = dm12i[idx] + l_sim_consts[mat_idx].d_t *
	    (- l_sim_consts[mat_idx].w12 * rho12r
             + OmRabi * inversion
	     - l_sim_consts[mat_idx].gamma12 * rho12i);

	/* real dm12 */
	dm12r_e = dm12r[idx] + l_sim_consts[mat_idx].d_t *
	    (+ l_sim_consts[mat_idx].w12 * rho12i
             - l_sim_consts[mat_idx].gamma12 * rho12r);

	real j = l_sim_consts[mat_idx].sigma * field;

	real p_t = l_sim_consts[mat_idx].M_CP * l_sim_consts[mat_idx].d12 *
	    (+ l_sim_consts[mat_idx].w12 * rho12i
             - l_sim_consts[mat_idx].gamma12 * rho12r);

	e_e = e[idx] + l_sim_consts[mat_idx].M_CE *
	    (-j - p_t + (h[idx + 1] - h[idx]) * l_sim_consts[mat_idx].d_x_inv);
    }

    /* apply sources */
    for (unsigned int k = 0; k < source_ct; k++) {
        int at = l_sim_sources[k].x_idx;
	if (gidx == at) {
            if (l_sim_sources[k].type == source::type::hard_source) {
                e_e = sd[l_sim_sources[k].data_base_idx + n];
            } else if (l_sim_sources[k].type == source::type::soft_source) {
                e_e += sd[l_sim_sources[k].data_base_idx + n];
            }
	}
    }

    ge[gidx] = e_e;
    d[gsize * 0 + gidx] = inv_e;
    d[gsize * 1 + gidx] = dm12i_e;
    d[gsize * 2 + gidx] = dm12r_e;

    /* copy results into scratchpad memory */
    for (unsigned int k = 0; k < copy_ct; k++) {
        if (l_copy_list[k].hasto_record(n)) {
            unsigned int pos = l_copy_list[k].get_position();
            record::type t = l_copy_list[k].get_type();
            real src_real;

            if (t == record::type::electric) {
                src_real = e_e;
            } else if (t == record::type::inversion) {
                src_real = inv_e;
            } else {
                /* TODO handle trouble, handle complex quantities */
            }

            if ((gidx >= pos) && (gidx < pos + l_copy_list[k].get_cols())) {
                int off_r = l_copy_list[k].get_offset_scratch_real(n,
                                                                   gidx - pos);
                scratch[off_r] = src_real;
            }
        }
    }
}

/* host members */
solver_cuda_2lvl_pc::solver_cuda_2lvl_pc(std::shared_ptr<const device> dev,
                                         std::shared_ptr<scenario> scen) :
solver_int(dev, scen)
{
    /* determine simulation settings */
    init_fdtd_simulation(dev, scen, 0.5);

    /* set up simulaton constants */
    std::map<std::string, unsigned int> id_to_idx;
    m_sim_consts = init_sim_constants(dev, scen, id_to_idx);

    /* TODO: sanity check scenario? */

    /* TODO: handle gridpoint number with %32 != 0 ? */
    if (scen->get_num_gridpoints() % 32 != 0) {
	throw std::invalid_argument("Number of grid points must be multiple"
				    " of 32");
    }

    if (dev->get_regions().size() == 0) {
        throw std::invalid_argument("No regions in device!");
    }

    if (m_sim_consts.size() >= MB_CUDA_MAX_MATERIALS) {
        throw std::invalid_argument("Too many materials in device!");
    }

    if (scen->get_sources().size() >= MB_CUDA_MAX_SOURCES) {
        throw std::invalid_argument("Too many sources in scenario!");
    }

    if (scen->get_records().size() >= MB_CUDA_MAX_CLE) {
        throw std::invalid_argument("Too many records requested in scenario!");
    }

    /* determine indices */
    unsigned int *mat_indices = new unsigned int[scen->get_num_gridpoints()];
    for (unsigned int i = 0; i < scen->get_num_gridpoints(); i++) {
        /* determine index of material */
        int idx = -1;
        real x = i * scen->get_gridpoint_size();
        for (const auto& reg : dev->get_regions()) {
            if ((x >= reg->get_start()) && (x <= reg->get_end())) {
                idx = id_to_idx[reg->get_material()->get_id()];
                break;
            }
        }
        /* TODO: assert/bug if idx == -1 */
        if ((idx < 0) || (idx >= dev->get_used_materials().size())) {
            throw std::invalid_argument("region not found");
        }
        mat_indices[i] = idx;
    }

    /* copy settings to CUDA constant memory */
    chk_err(hipMemcpyToSymbol(HIP_SYMBOL(l_sim_consts), m_sim_consts.data(),
                               m_sim_consts.size() *
                               sizeof(sim_constants_2lvl)));

    /* allocate buffers on GPU */
    chk_err(hipMalloc(&m_e, sizeof(real) * scen->get_num_gridpoints()));
    chk_err(hipMalloc(&m_h, sizeof(real) * (scen->get_num_gridpoints() + 1)));
    chk_err(hipMalloc(&m_d, sizeof(real) * scen->get_num_gridpoints() * 3));
    chk_err(hipMalloc(&m_mat_indices, sizeof(unsigned int) *
                       scen->get_num_gridpoints()));

    /* copy indices to GPU */
    chk_err(hipMemcpy(m_mat_indices, mat_indices, sizeof(unsigned int) *
                       scen->get_num_gridpoints(), hipMemcpyHostToDevice));

    delete[] mat_indices;

    /* initialize memory */
    /* TODO move to class member */
    unsigned int threads = 128;
    unsigned int blocks = scen->get_num_gridpoints()/threads;

    init_memory<<<blocks, threads>>>(m_d, m_e, m_h, m_mat_indices);

    /* set up results and transfer data structures */
    unsigned int scratch_size = 0;
    for (const auto& rec : scen->get_records()) {
        /* create copy list entry */
        copy_list_entry entry(rec, scen, scratch_size);

        /* add result to solver */
        m_results.push_back(entry.get_result());

        /* calculate scratch size */
        scratch_size += entry.get_size();

       /* take imaginary part into account */
        if (rec->is_complex()) {
            scratch_size += entry.get_size();
        }

        /* TODO check if result is available */
        /*
           throw std::invalid_argument("Requested result is not available!");
        */

        m_copy_list.push_back(entry);
    }

    /* allocate scratchpad result memory */
    chk_err(hipMalloc(&m_result_scratch, sizeof(real) * scratch_size));

    /* set up sources */
    unsigned int source_data_size = scen->get_num_timesteps() *
        scen->get_sources().size();
    chk_err(hipMalloc(&m_source_data, sizeof(real) * source_data_size));
    real *source_data = new real[source_data_size];
    unsigned int base_idx = 0;
    for (const auto& src : scen->get_sources()) {
        sim_source s;
        s.type = src->get_type();
        s.x_idx = src->get_position()/scen->get_gridpoint_size();
        s.data_base_idx = base_idx;
        m_sim_sources.push_back(s);

        /* calculate source values */
        for (unsigned int j = 0; j < scen->get_num_timesteps(); j++) {
            source_data[base_idx + j] =
                src->get_value(j * scen->get_timestep_size());
        }

        base_idx += scen->get_num_timesteps();
    }

    /* copy indices to GPU */
    chk_err(hipMemcpy(m_source_data, source_data, sizeof(unsigned int) *
                       source_data_size, hipMemcpyHostToDevice));
    delete[] source_data;

    /* copy source properties to GPU constant memory */
    chk_err(hipMemcpyToSymbol(HIP_SYMBOL(l_sim_sources), m_sim_sources.data(),
                               m_sim_sources.size() *
                               sizeof(sim_source)));

    /* copy copy list entries to GPU constant memory */
    for (unsigned int i = 0; i < m_copy_list.size(); i++) {
        chk_err(hipMemcpyToSymbol(HIP_SYMBOL(l_copy_list), &m_copy_list[i].get_dev(),
                                   sizeof(copy_list_entry_dev),
                                   sizeof(copy_list_entry_dev) * i));

    }

}

solver_cuda_2lvl_pc::~solver_cuda_2lvl_pc()
{
    /* free CUDA memory */
    hipFree(m_h);
    hipFree(m_e);
    hipFree(m_d);
    hipFree(m_mat_indices);
    hipFree(m_result_scratch);
    hipFree(m_source_data);

    /* reset device */
    hipDeviceReset();
}

const std::string&
solver_cuda_2lvl_pc::get_name() const
{
    return factory.get_name();
}

void
solver_cuda_2lvl_pc::run() const
{
    unsigned int threads = 128;
    unsigned int blocks = m_scenario->get_num_gridpoints()/threads;
    /* TODO handle roundoff errors in thread/block partition */

    /* main loop */
    for (unsigned int i = 0; i < m_scenario->get_num_timesteps(); i++) {

	/* makestep e and density matrix */
	makestep_e_dm<<<blocks, threads,
	    (6 * threads + 1) * sizeof(real)>>>(m_d, m_h, m_e, m_mat_indices,
                                                m_result_scratch,
                                                m_source_data,
                                                m_sim_sources.size(),
                                                m_copy_list.size(),
                                                i);

	/* makestep h */
	makestep_h<<<blocks, threads, (threads + 1) * sizeof(real)>>>
	    (m_e, m_h, m_mat_indices);

    }

    /* bulk copy results into result classes */
    for (const auto& cle : m_copy_list) {
        chk_err(hipMemcpy(cle.get_result()->get_data_real_raw(),
                           m_result_scratch +
                           cle.get_offset_scratch_real(0, 0),
                           cle.get_size() * sizeof(real),
                           hipMemcpyDeviceToHost));
        if (cle.is_complex()) {
            /*
            chk_err(hipMemcpy(cle.get_result_imag(0, 0).data(),
                               cle.get_scratch_imag(0, 0), cle.get_size(),
                               hipMemcpyDeviceToHost));
            */
        }
    }
}

}
