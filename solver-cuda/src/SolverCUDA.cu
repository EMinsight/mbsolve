#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <SolverCUDA.hpp>

namespace mbsolve {

__global__ void makestep_black()
{

}

SolverCUDA::SolverCUDA() : Solver("CUDA Solver")
{
}

SolverCUDA::~SolverCUDA()
{
}

void SolverCUDA::do_setup(const Device& device, const Scenario& scenario)
{
}

void SolverCUDA::do_cleanup()
{
}

void SolverCUDA::do_run()
{
    makestep_black<<<12, 1>>>();
}

}
