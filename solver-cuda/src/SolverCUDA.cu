#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <SolverCUDA.hpp>

namespace mbsolve {

__global__ void makestep_black()
{

}

SolverCUDA::SolverCUDA() : Solver("CUDA Solver")
{
}

SolverCUDA::~SolverCUDA()
{
}

void SolverCUDA::setup()
{
}

void SolverCUDA::cleanup()
{
}

void SolverCUDA::run()
{
    makestep_black<<<12, 1>>>();
}

}
