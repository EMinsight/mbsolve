#include "hip/hip_runtime.h"
#include <boost/foreach.hpp>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <SolverCUDA2lvl.hpp>

namespace mbsolve {

static SolverFactory<SolverCUDA2lvl> factory("cuda-2lvl");

static inline void chk_err(hipError_t code)
{
    if (code != hipSuccess) {
	throw std::runtime_error(std::string("CUDA: ") +
				 hipGetErrorString(code));
    }
}

/* CUDA memory and kernels */
__device__ __constant__ struct sim_constants gsc[MaxRegions];

__device__ __inline__ unsigned int get_region(unsigned int idx)
{
    for (unsigned int i = 0; i < MaxRegions; i++) {
	if (idx < gsc[i].idx_end) {
	    return i;
	}
    }
    return 0;
}

__global__ void init_memory(const DensityMatrix& dm, real *e, real *h)
{
    unsigned int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int max = blockDim.x * gridDim.x - 1;

    /* TODO: alternative initializations */
    hiprandState_t rand_state;

    /* initialize random number generator */
    hiprand_init(clock64(), gidx, 0, &rand_state);

    if (gidx == max - 1) {
	h[max] = 0.0;
    }
    h[gidx] = 0.0;
    e[gidx] = hiprand_uniform(&rand_state) * 1e-15;

    real populations[NumLevels];
    real trace = 0.0;

    for (unsigned int i = 0; i < NumLevels; i++) {
	populations[i] = hiprand_uniform(&rand_state);
	trace += populations[i];
    }

    for (unsigned int row = 0; row < NumLevels; row++) {
	for (unsigned int col = 0; col < NumLevels; col++) {
	    if (row == col) {
		if (row == NumLevels - 1) {
		    dm.OldDM(row, col)[gidx] = populations[row]/trace;
		} else {
		    dm.OldDM(row, col)[gidx] = 0.0;
		}
		for (int i = 0; i < NumMultistep; i++) {
		    dm.RHS(row, col, i)[gidx] = 0.0;
		}
	    }
	}
    }
}

__global__ void makestep_h(const real *ge, real *gh)
{
    int idx = threadIdx.x;
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int region = get_region(gidx);

    extern __shared__ real e[];

    if ((idx == 0) && (gidx != 0)) {
	e[0] = ge[gidx - 1];
    }
    e[idx + 1] = ge[gidx];

    __syncthreads();

    /* TODO: alternative boundary conditions? */
    /* TODO: different kernel or templated version?? */
    /* open circuit boundary conditions already set */
    /* gh_ghz[0] = 0; */
    /* gh_ghz[N_x] = 0; */

    if (gidx != 0) {
	gh[gidx] += gsc[region].M_CH * (e[idx + 1] - e[idx]);
    }
}

__global__ void makestep_e(const DensityMatrix& dm, const real *gh, real *ge)
{
    int idx = threadIdx.x;
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int region = get_region(gidx);

    extern __shared__ real h[];

    h[idx] = gh[gidx];
    if (idx == blockDim.x - 1) {
	h[idx + 1] = gh[gidx + 1];
    }

    __syncthreads();

    real j = ge[gidx] * gsc[region].sigma;
    real p_t = gsc[region].M_CP * gsc[region].d12 * dm.OldDM(0, 1)[gidx];

    ge[gidx] += gsc[region].M_CE *
	(-j - p_t + (h[idx + 1] - h[idx])/gsc[region].d_x);
}

__global__ void makestep_dm(const DensityMatrix& dm, const real *ge)
{
    //    int idx = threadIdx.x;
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int region = get_region(gidx);
    int row = blockIdx.y;
    int col = blockIdx.z;

    real rhs = 0.0;

    if ((row == 0) && (col == 0)) {
	/* dm11 */
	rhs = - dm.OldDM(0, 1)[gidx] * 2.0 * gsc[region].d12 * ge[gidx]
	    - dm.OldDM(0, 0)[gidx] * gsc[region].tau1;
    } else if ((row == 0) && (col == 1)) {
	/* imag dm12 */
	rhs = dm.OldDM(1, 0)[gidx] * gsc[region].w12
	    - dm.OldDM(0, 1)[gidx] * gsc[region].gamma12;
    } else if ((row == 1) && (col == 0)) {
	/* real dm12 */
	rhs = - dm.OldDM(0, 1)[gidx] * gsc[region].w12
	    - dm.OldDM(1, 0)[gidx] * gsc[region].gamma12;
    } else if ((row == 1) && (col == 1)) {
	/* dm22 */
	rhs = dm.OldDM(0, 1)[gidx] * 2.0 * gsc[region].d12 * ge[gidx]
	    + dm.OldDM(0, 0)[gidx] * gsc[region].tau1;
    } else {
	/* do nothing */
    }

    dm.RHS(row, col, 0)[gidx] = rhs;
    dm.NewDM(row, col)[gidx] = dm.OldDM(row, col)[gidx] + gsc[region].d_t *
	(+ dm.RHS(row, col, 0)[gidx] * 1901.0/720.0
	 - dm.RHS(row, col, 1)[gidx] * 1387.0/360.0
	 + dm.RHS(row, col, 2)[gidx] * 109.0/30.0
	 - dm.RHS(row, col, 3)[gidx] * 637.0/360.0
	 + dm.RHS(row, col, 4)[gidx] * 251.0/720.0);
}


DensityMatrix::DensityMatrix() : a_is_old(true), head(0)
{
}

DensityMatrix::~DensityMatrix()
{
    for (unsigned int i = 0; i < NumLevels; i++) {
	for (unsigned int j = 0; j < NumLevels; j++) {
	    hipFree(dm_a[i][j]);
	    hipFree(dm_b[i][j]);
	    for (unsigned int k = 0; k < NumMultistep; k++) {
		hipFree(rhs[i][j][k]);
	    }
	}
    }
}

__device__ __inline__ real *
DensityMatrix::OldDM(unsigned int row, unsigned int col) const
{
    return a_is_old ? dm_a[row][col] : dm_b[row][col];
}

__device__ __inline__ real *
DensityMatrix::NewDM(unsigned int row, unsigned int col) const
{
    return a_is_old ? dm_b[row][col] : dm_a[row][col];
}

__device__ __inline__ real *
DensityMatrix::RHS(unsigned int row, unsigned int col,
		   unsigned int rhsIdx) const
{
    return rhs[row][col][(rhsIdx + head) % NumMultistep];
}

void
DensityMatrix::next()
{
    a_is_old = !a_is_old;
    head = (head + 1) % NumMultistep;
}

void
DensityMatrix::initialize(unsigned int numGridPoints)
{
    for (unsigned int i = 0; i < NumLevels; i++) {
	for (unsigned int j = 0; j < NumLevels; j++) {
	    chk_err(hipMalloc(&dm_a[i][j], sizeof(real) * numGridPoints));
	    chk_err(hipMalloc(&dm_b[i][j], sizeof(real) * numGridPoints));
	    for (unsigned int k = 0; k < NumMultistep; k++) {
		chk_err(hipMalloc(&rhs[i][j][k],
				   sizeof(real) * numGridPoints));
	    }
	}
    }
}

/* host members */
SolverCUDA2lvl::SolverCUDA2lvl(const Device& device,
			       const Scenario& scenario) :
    ISolver(device, scenario), comp_maxwell(0), comp_density(0), copy(0)
{
    /* total device length */
    Quantity length = device.XDim();

    /* minimum relative permittivity */
    Quantity minRelPermittivity = device.MinRelPermittivity();

    /* TODO: sanity check scenario? */
    if (m_scenario.NumGridPoints % 32 != 0) {
	throw std::invalid_argument("Number of grid points must be multiple"
				    " of 32");
    }

    /* determine grid point and time step size */
    real C = 0.9; /* courant number */
    real velocity = sqrt(MU0() * EPS0() * minRelPermittivity());
    m_scenario.GridPointSize = length()/(m_scenario.NumGridPoints - 1);
    real timestep  = C * m_scenario.GridPointSize * velocity;
    m_scenario.NumTimeSteps = ceil(m_scenario.SimEndTime/timestep) + 1;
    m_scenario.TimeStepSize = m_scenario.SimEndTime /
	(m_scenario.NumTimeSteps - 1);


    /* determine border indices and initialize region settings */
    if (device.Regions.size() > MaxRegions) {
	throw std::invalid_argument("Too many regions requested");
    }
    struct sim_constants sc[MaxRegions];

    unsigned int i;
    BOOST_FOREACH(Region reg, device.Regions) {
	if (i > 0) {
	    sc[i - 1].idx_end = round(reg.X0()/m_scenario.GridPointSize) - 1;
	}
	sc[i].idx_start = round(reg.X0()/m_scenario.GridPointSize);
	sc[i].M_CE = m_scenario.TimeStepSize/(EPS0() * reg.RelPermittivity());
	sc[i].M_CH = m_scenario.TimeStepSize/(MU0() *
					      m_scenario.GridPointSize);
	sc[i].M_CP = -2.0 * reg.DopingDensity * E0;
	sc[i].sigma = 2.0 * sqrt(EPS0 * reg.RelPermittivity/MU0) * reg.Losses;

	sc[i].w12 = (reg.TransitionFrequencies.size() < 1) ? 0.0 :
	    reg.TransitionFrequencies[0]();
	/* TODO rename to rabi freqs or something */
	sc[i].d12 = (reg.DipoleMoments.size() < 1) ? 0.0 :
	    reg.DipoleMoments[0]() * E0()/HBAR();
	sc[i].tau1 = (reg.ScatteringRates.size() < 1) ? 0.0 :
	    reg.ScatteringRates[0]();
	sc[i].gamma12 = (reg.DephasingRates.size() < 1) ? 0.0 :
	    reg.DephasingRates[0]();

	sc[i].d_x = m_scenario.GridPointSize;
	sc[i].d_t = m_scenario.TimeStepSize;
	i++;
    }
    if (i > 0) {
	sc[i - 1].idx_end = m_scenario.NumGridPoints - 1;
    }

    /* initialize streams */
    chk_err(hipStreamCreate(&comp_maxwell));
    chk_err(hipStreamCreate(&comp_density));
    chk_err(hipStreamCreate(&copy));

    /* allocate space */
    chk_err(hipMalloc(&e, sizeof(real) * m_scenario.NumGridPoints));
    chk_err(hipMalloc(&h, sizeof(real) * (m_scenario.NumGridPoints + 1)));
    dm.initialize(m_scenario.NumGridPoints);

    /* initalize memory */
    /* TODO: kernel call */

    /* copy settings to CUDA constant memory */
    chk_err(hipMemcpyToSymbol(HIP_SYMBOL(gsc), &sc, MaxRegions *
			       sizeof(struct sim_constants)));

    /* set up results transfer data structures */
    BOOST_FOREACH(Record rec, m_scenario.Records) {
	unsigned int interval = ceil(rec.Interval()/m_scenario.TimeStepSize);
	unsigned int row_ct = m_scenario.NumTimeSteps/interval;
	unsigned int position_idx;
	unsigned int col_ct;

	if (rec.Position() < 0.0) {
	    /* copy complete grid */
	    position_idx = 0;
	    col_ct = m_scenario.NumGridPoints;
	} else {
	    position_idx = round(rec.Position()/m_scenario.GridPointSize);
	    col_ct = 1;
	}

	/* allocate result memory */
	Result *res = new Result(rec.Name, col_ct, row_ct);
	m_results.push_back(res);

	/* create copy list entry */
	CopyListEntry *entry;
	if (rec.Type == EField) {
	    entry = new CLEField(e, res, col_ct * row_ct, position_idx,
				 interval);
	    m_copyListRed.push_back(entry);
	} else if (rec.Type == HField) {
	    entry = new CLEField(h, res, col_ct * row_ct, position_idx,
				 interval);
	    m_copyListBlack.push_back(entry);
	} else if (rec.Type == Density) {
	    if ((rec.I - 1 < 2) && (rec.J - 1 < 2)) {
		if (rec.I == rec.J) {
		    /* main diagonal entry */
		    entry = new CLEDensity(&dm, rec.I, rec.J, res,
					   col_ct * row_ct, position_idx,
					   interval);
		    m_copyListBlack.push_back(entry);
		} else {
		    /* off-diagonal entry */
		    /* TODO */
		    /* if complex */
		    /* create two list entries */
		    /* create two Results, or one complex Result */

		    /* real part: GetSrcDensity(&dm, rec.I, rec.J); */
		    /* imag part: GetSrcDensity(&dm, rec.J, rec.I); */
		}
	    } else {
	    // throw exc
	    }
	} else {
	    // throw exc
	}
    }
}

SolverCUDA2lvl::~SolverCUDA2lvl()
{
    /* delete copy lists */
    BOOST_FOREACH(CopyListEntry *entry, m_copyListRed) {
	delete entry;
    }
    BOOST_FOREACH(CopyListEntry *entry, m_copyListBlack) {
	delete entry;
    }

    /* delete results */
    BOOST_FOREACH(Result *result, m_results) {
	delete result;
    }

    /* free CUDA memory */
    hipFree(h);
    hipFree(e);

    /* clean up streams */
    if (comp_maxwell) {
	hipStreamDestroy(comp_maxwell);
    }
    if (comp_density) {
	hipStreamDestroy(comp_density);
    }
    if (copy) {
	hipStreamDestroy(copy);
    }

    /* reset device */
    hipDeviceReset();
}

std::string
SolverCUDA2lvl::getName() const
{
    return std::string("CUDA two-level solver");
}

void
SolverCUDA2lvl::run(const std::vector<Result *>& results) const
{
    unsigned int threads = 128;
    unsigned int blocks = 10; // NumGridPoint / threads
    /* TODO handle roundoff errors in thread/block partition */

    dim3 block(blocks);
    dim3 thread(threads);

    /* main loop */
    for (unsigned int i = 1; i < m_scenario.NumTimeSteps; i++) {
	/* makestep_h in maxwell stream */
	/* makestep_dm in density stream */
	/* gather e field in copy stream */
	BOOST_FOREACH(CopyListEntry *entry, m_copyListRed) {
	    if (entry->record(i)) {
		hipMemcpyAsync(entry->getDst(i), entry->getSrc(),
				entry->getSize(), hipMemcpyDeviceToHost,
				copy);
	    }
	}


	/* sync */

	/* call toggle */

	/* calculate source value -> makestep_e kernel */

	/* gather h field and dm entries in copy stream */
	BOOST_FOREACH(CopyListEntry *entry, m_copyListBlack) {
	    if (entry->record(i)) {
		hipMemcpyAsync(entry->getDst(i), entry->getSrc(),
				entry->getSize(), hipMemcpyDeviceToHost,
				copy);
	    }
	}


	/* makestep_e */
	/* sync */

    }

    //    makestep_h<<<block, thread, sizeof(real) * (threads + 1)>>>(e, h);

}

}
