#include <CUDACommon.hpp>
#include <CUDADensityMatrix.hpp>

namespace mbsolve {

__host__
CUDADensityMatrixData::CUDADensityMatrixData(unsigned int numGridPoints,
					     unsigned int numLevels,
					     unsigned int numMultistep,
					     real *data) :
    m_numGridPoints(numGridPoints), m_numLevels(numLevels),
    m_numMultistep(numMultistep), m_aIsOld(false), m_head(0),
    m_sizeData(sizeof(real) * numLevels * numLevels * (2 + numMultistep) *
	       numGridPoints),
    m_dmA(data), m_dmB(data + numLevels * numLevels * numGridPoints),
    m_rhs(data + numLevels * numLevels * 2 * numGridPoints)
{
}

/*
__host__ __device__
CUDADensityMatrixData::CUDADensityMatrixData(const CUDADensityMatrixData&
					     other) :
    m_numLevels(other.m_numLevels), m_numMultistep(other.m_numMultistep),
    m_sizeData(other.m_sizeData), m_dmA(other.m_dmA), m_dmB(other.m_dmB),
    m_rhs(other.m_rhs), m_head(other.m_head), m_aIsOld(other.m_aIsOld)
{
}*/


__host__
CUDADensityMatrixData::~CUDADensityMatrixData()
{
}

__host__ unsigned int
CUDADensityMatrixData::getSize() const
{
    return m_sizeData;
}

__host__ __device__ unsigned int
CUDADensityMatrixData::getNumLevels() const
{
    return m_numLevels;
}

__host__ __device__ unsigned int
CUDADensityMatrixData::getNumMultistep() const
{
    return m_numMultistep;
}

__host__ __device__ real *
CUDADensityMatrixData::oldDM(unsigned int row, unsigned int col) const
{
    unsigned int i = (row * m_numLevels + col) * m_numGridPoints;
    return (m_aIsOld ? m_dmA : m_dmB) + i;
}

__host__ __device__ real *
CUDADensityMatrixData::newDM(unsigned int row, unsigned int col) const
{
    unsigned int i = (row * m_numLevels + col) * m_numGridPoints;
    return (m_aIsOld ? m_dmB : m_dmA) + i;
}

__host__ __device__ real *
CUDADensityMatrixData::rhs(unsigned int row, unsigned int col,
			   unsigned int rhsIdx) const
{
    unsigned int idx = (rhsIdx + m_head) % m_numMultistep;
    unsigned int base = (row * m_numLevels + col) * m_numMultistep;
    return m_rhs + (base + idx) * m_numGridPoints;
}

__host__ __device__ void
CUDADensityMatrixData::next()
{
    m_aIsOld = !m_aIsOld;
    m_head = (m_head + 1) % m_numMultistep;
}



CUDADensityMatrix::CUDADensityMatrix(unsigned int numGridPoints,
				     unsigned int numLevels,
				     unsigned int numMultistep) :
    m_data(numGridPoints, numLevels, numMultistep)
    // m_numLevels(numLevels),
    //m_numMultistep(numMultistep)
{
    /* allocate GPU memory */
    chk_err(hipMalloc(&m_gpuBuffer, m_data.getSize()));

    m_data = CUDADensityMatrixData(numGridPoints, numLevels, numMultistep,
				   m_gpuBuffer);
}

CUDADensityMatrix::~CUDADensityMatrix()
{
    hipFree(m_gpuBuffer);
}

CUDADensityMatrixData&
CUDADensityMatrix::getData()
{
    return m_data;
}
/*
const CUDADensityMatrixData&
CUDADensityMatrix::getHostData() const
{
    return m_data;
}

void
CUDADensityMatrix::next()
{
    m_data.next();
    }*/

}
